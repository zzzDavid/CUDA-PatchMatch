#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"
__global__ void helloFromGPU(void)
{
	printf("Hello World from GPU!\n");
}

int main(void)
{
	// hello from cpu
	printf("Hello World from CPU!\n");

	helloFromGPU << <1, 10 >> > ();

	hipDeviceReset();

	return 0;
}